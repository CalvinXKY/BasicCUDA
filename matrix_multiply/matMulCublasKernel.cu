#include "matMul.h"
#include <hipblas.h>


/**
 * Run a simple test of matrix multiplication using CUBLAS Sgemm.
 */
int MatMulCublasTest(int argc, char **argv, int blockSize, int iterNum, const dim3 &dimsA, const dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
    hipStream_t stream;

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasHandle_t handle;
    checkCudaErrors(hipblasCreate(&handle));
    
    // Create and start timer
    printf("Computing result using CUBLAS Sgemmm Kernel. \n");
    checkCudaErrors(hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimsB.x, dimsA.y,
        dimsA.x, &alpha, d_B, dimsB.x, d_A,
        dimsA.x, &beta, d_C, dimsB.x));

    printf("Warmup  operation done\n");
    checkCudaErrors(hipStreamSynchronize(stream));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, stream));

    // Execute the kernel
    for (int j = 0; j < iterNum; j++) {
      // note cublas is column primary!
      // need to transpose the order
      checkCudaErrors(hipblasSgemm(
          handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimsB.x, dimsA.y,
          dimsA.x, &alpha, d_B, dimsB.x, d_A,
          dimsA.x, &beta, d_C, dimsB.x));
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / iterNum;
    double flopsPerMatrixMul =
        2.0 * static_cast<double>(dimsA.x) * static_cast<double>(dimsA.y) * static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,",
           gigaFlops, msecPerMatrixMul, flopsPerMatrixMul);

    // Copy result from device to host
    checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipblasDestroy(handle));

    bool ret = ResultCheck(h_C, static_cast<int>(dimsC.x * dimsC.y), dimsA.x, valB);

    // Clean up memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipStreamDestroy(stream));

    if (ret) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}
