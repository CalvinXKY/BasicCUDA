#include "hip/hip_runtime.h"


#include "matMul.h"

template <int BLOCK_SIZE> __global__ void MatMulKernel2DAnySize(float *C, float *A, float *B, int wA, int wC, int hC)
{
    int wB = wC;
    int maxIdxA = wA * hC;

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    while (wA * BLOCK_SIZE * by < maxIdxA) {
        // Index of the first sub-matrix of A processed by the block
        int aBegin = wA * BLOCK_SIZE * by;

        // Index of the last sub-matrix of A processed by the block
        int aEnd = aBegin + wA - 1;

        // Step size used to iterate through the sub-matrices of A
        int aStep = BLOCK_SIZE;

        // Index of the first sub-matrix of B processed by the block
        int bBegin = BLOCK_SIZE * bx;

        // Step size used to iterate through the sub-matrices of B
        int bStep = BLOCK_SIZE * wB;

        // Csub is used to store the element of the block sub-matrix
        // that is computed by the thread
        float Csub = 0;

        // Loop over all the sub-matrices of A and B
        // required to compute the block sub-matrix
        int flag = 0;
        for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
            // Declaration of the shared memory array As used to
            // store the sub-matrix of A
            __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

            // Declaration of the shared memory array Bs used to
            // store the sub-matrix of B
            __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

            // Load the matrices from device memory
            // to shared memory; each thread loads
            // one element of each matrix
            if (flag * BLOCK_SIZE + tx < wA || flag * BLOCK_SIZE + ty < hC) {
                As[ty][tx] = A[a + wA * ty + tx];
            } else {
                As[ty][tx] = 0.0;
            }

            if (flag * BLOCK_SIZE + ty < wA || flag * BLOCK_SIZE + tx < wC) {
                Bs[ty][tx] = B[b + wB * ty + tx];
            } else {
                Bs[ty][tx] = 0.0;
            }

            // Bs[ty][tx] = B[idx];

            // Bs[ty][tx] = B[idx];
            // Synchronize to make sure the matrices are loaded
            __syncthreads();

            // Multiply the two matrices together;
            // each thread computes one element
            // of the block sub-matrix
#pragma unroll

            for (int k = 0; k < BLOCK_SIZE; ++k) {
                Csub += As[ty][k] * Bs[k][tx];
            }

            // Synchronize to make sure that the preceding
            // computation is done before loading two new
            // sub-matrices of A and B in the next iteration
            __syncthreads();
            flag++;
        }

        // Write the block sub-matrix to device memory;
        // each thread writes one element
        if (BLOCK_SIZE * bx + tx < wC && BLOCK_SIZE * by + ty < hC) { // thread could over max.
            C[wB * BLOCK_SIZE * by + BLOCK_SIZE * bx + wB * ty + tx] = Csub;
        }
        bx += BLOCK_SIZE;
        by += BLOCK_SIZE;
    }
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatMulKernel2DBlockMultiplesSize(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

/**
 * Run a simple test of matrix multiplication with 2D blocks.
 */
int MatMul2DTest(int argc, char **argv, int blockSize, int iterNum, const dim3 &dimsA, const dim3 &dimsB,
                 bool useAnySize)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
    hipStream_t stream;

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

    // Setup execution parameters
    dim3 threads(blockSize, blockSize);
    dim3 grid;

    // Create and start timer
    printf("Computing result using MatMul2DTest Kernel. \n");
    if (useAnySize)
        printf("Spport any size, e.g. wA=1000 hA=312 wB=11 hB=1000.\n");

    // select diff blocks for kerenl
    void (*MMKernel2DAnySizeExe)(float *, float *, float *, int, int, int);
    void (*MMKernel2DFixSizeExe)(float *, float *, float *, int, int);
    if (blockSize <= 16) {
        MMKernel2DFixSizeExe = MatMulKernel2DBlockMultiplesSize<16>;
        MMKernel2DAnySizeExe = MatMulKernel2DAnySize<16>;
    } else {
        MMKernel2DFixSizeExe = MatMulKernel2DBlockMultiplesSize<32>;
        MMKernel2DAnySizeExe = MatMulKernel2DAnySize<32>;
    }

    // Performs warmup operation using matrixMul CUDA kernel
    if (useAnySize) {
        grid = dim3(dimsB.x / threads.x + 1, dimsA.y / threads.y + 1);
        MMKernel2DAnySizeExe<<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsC.x, dimsC.y);
    } else {
        grid = dim3(dimsB.x / threads.x, dimsA.y / threads.y);
        MMKernel2DFixSizeExe<<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("Warmup  operation done\n");
    checkCudaErrors(hipStreamSynchronize(stream));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, stream));

    // Execute the kernel
    for (int j = 0; j < iterNum; j++) {
        if (useAnySize) {
            MMKernel2DAnySizeExe<<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsC.x, dimsC.y);
        } else {
            MMKernel2DFixSizeExe<<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / iterNum;
    double flopsPerMatrixMul =
        2.0 * static_cast<double>(dimsA.x) * static_cast<double>(dimsA.y) * static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
           " WorkgroupSize= %u threads/block\n",
           gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    bool ret = ResultCheck(h_C, static_cast<int>(dimsC.x * dimsC.y), dimsA.x, valB);

    // Clean up memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipStreamDestroy(stream));

    if (ret) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}
