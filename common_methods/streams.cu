
#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


template <typename T> void check(T result, char const *const func, const char *const file, int const line)
{
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(result),
                hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)


#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*200)

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

void singleStream(bool isAsync) {
        hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // start the timers
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // initialize the stream
    checkCudaErrors(hipStreamCreate(&stream));

    // allocate the memory on the GPU
    checkCudaErrors(hipMalloc((void**)&dev_a,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_b,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_c,
                              N * sizeof(int)));

    // allocate host locked memory, used to stream
    checkCudaErrors(hipHostAlloc((void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault));

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    checkCudaErrors(hipEventRecord(start, 0));
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N) {

        if (isAsync) {
            // copy the locked memory to the device;
            checkCudaErrors(hipMemcpy(dev_a, host_a+i,
                                    N * sizeof(int),
                                    hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_b, host_b+i,
                                    N * sizeof(int),
                                    hipMemcpyHostToDevice));

            kernel<<<N/256,256,0,stream>>>(dev_a, dev_b, dev_c);

            // copy the data from device to locked memory
            checkCudaErrors(hipMemcpy(host_c+i, dev_c,
                                    N * sizeof(int),
                                    hipMemcpyDeviceToHost));
        } else {
            // copy the locked memory to the device, async
            checkCudaErrors(hipMemcpyAsync(dev_a, host_a+i,
                                        N * sizeof(int),
                                        hipMemcpyHostToDevice,
                                        stream));
            checkCudaErrors(hipMemcpyAsync(dev_b, host_b+i,
                                        N * sizeof(int),
                                        hipMemcpyHostToDevice,
                                        stream));

            kernel<<<N/256,256,0,stream>>>(dev_a, dev_b, dev_c);

            // copy the data from device to locked memory
            checkCudaErrors(hipMemcpyAsync(host_c+i, dev_c,
                                        N * sizeof(int),
                                        hipMemcpyDeviceToHost,
                                        stream));
        }


    }
    // copy result chunk from locked to full buffer
    checkCudaErrors(hipStreamSynchronize(stream));

    checkCudaErrors(hipEventRecord(stop, 0));

    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime,
                                        start, stop));
    if (isAsync) {
        printf("Single stream with async copy. Elapsed time:  %3.1f ms\n", elapsedTime);
    } else {
        printf("Single stream. Elapsed time:  %3.1f ms\n", elapsedTime);
    }

    // cleanup the streams and memory
    checkCudaErrors(hipHostFree(host_a));
    checkCudaErrors(hipHostFree(host_b));
    checkCudaErrors(hipHostFree(host_c));
    checkCudaErrors(hipFree(dev_a));
    checkCudaErrors(hipFree(dev_b));
    checkCudaErrors(hipFree(dev_c));
    checkCudaErrors(hipStreamDestroy(stream));
}

void doubleStreamAsyncCopy() {
    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    // start the timers
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // initialize the streams
    checkCudaErrors(hipStreamCreate(&stream0));
    checkCudaErrors(hipStreamCreate(&stream1));

    // allocate the memory on the GPU
    checkCudaErrors(hipMalloc((void**)&dev_a0,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_b0,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_c0,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_a1,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_b1,
                              N * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_c1,
                              N * sizeof(int)));

    // allocate host locked memory, used to stream
    checkCudaErrors(hipHostAlloc((void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault));

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    checkCudaErrors(hipEventRecord(start, 0));
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N*2) {
        // enqueue copies of a in stream0 and stream1
        checkCudaErrors(hipMemcpyAsync(dev_a0, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0));
        checkCudaErrors(hipMemcpyAsync(dev_a1, host_a+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1));
        // enqueue copies of b in stream0 and stream1
        checkCudaErrors(hipMemcpyAsync(dev_b0, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0));
        checkCudaErrors(hipMemcpyAsync(dev_b1, host_b+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1));

        // enqueue kernels in stream0 and stream1
        kernel<<<N/256,256,0,stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N/256,256,0,stream1>>>(dev_a1, dev_b1, dev_c1);

        // enqueue copies of c from device to locked memory
        checkCudaErrors(hipMemcpyAsync(host_c+i, dev_c0,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream0));
        checkCudaErrors(hipMemcpyAsync(host_c+i+N, dev_c1,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream1));
    }
    checkCudaErrors(hipStreamSynchronize(stream0));
    checkCudaErrors(hipStreamSynchronize(stream1));

    checkCudaErrors(hipEventRecord(stop, 0));

    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime,
                                        start, stop));
    printf("Double stream with async copy. Elapsed time:  %3.1f ms\n", elapsedTime);

    // cleanup the streams and memory
    checkCudaErrors(hipHostFree(host_a));
    checkCudaErrors(hipHostFree(host_b));
    checkCudaErrors(hipHostFree(host_c));
    checkCudaErrors(hipFree(dev_a0));
    checkCudaErrors(hipFree(dev_b0));
    checkCudaErrors(hipFree(dev_c0));
    checkCudaErrors(hipFree(dev_a1));
    checkCudaErrors(hipFree(dev_b1));
    checkCudaErrors(hipFree(dev_c1));
    checkCudaErrors(hipStreamDestroy(stream0));
    checkCudaErrors(hipStreamDestroy(stream1));
}

int main(void) {
    hipDeviceProp_t  prop;
    int whichDevice;
    checkCudaErrors(hipGetDevice(&whichDevice));
    checkCudaErrors(hipGetDeviceProperties(&prop, whichDevice));
    if (prop.asyncEngineCount < 1) {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    } else {
        printf("The asyncEngineCount in device is:%d\n", prop.asyncEngineCount);
    }
    singleStream(false);
    singleStream(true);
    doubleStreamAsyncCopy();
    return 0;
}
